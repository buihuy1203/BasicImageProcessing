#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>

using namespace cv;
using namespace std;

__global__ void brightnessKernel(const uchar *input, uchar *output, int rows, int cols, int bright) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < rows && x < cols) {
        int idx = (y * cols + x) * 3;
        output[idx] = min(max(input[idx] + bright, 0), 255);         // Blue
        output[idx + 1] = min(max(input[idx + 1] + bright, 0), 255); // Green
        output[idx + 2] = min(max(input[idx + 2] + bright, 0), 255); // Red
    }
}

Mat ParallelBrightnessCUDA(Mat &input, int bright) {
    // Image size
    int rows = input.rows;
    int cols = input.cols;

    // Input and output data
    size_t dataSize = rows * cols * 3 * sizeof(uchar);
    uchar *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);

    // Copy input data to device
    hipMemcpy(d_input, input.data, dataSize, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    brightnessKernel<<<gridSize, blockSize>>>(d_input, d_output, rows, cols, bright);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy output data back to host
    Mat result(input.size(), CV_8UC3);
    hipMemcpy(result.data, d_output, dataSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return result;
}
