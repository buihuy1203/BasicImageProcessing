#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>

using namespace cv;
using namespace std;

__global__ void yCrCBKernel(const uchar *input, uchar *output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < rows && x < cols) {
        int idx = (y * cols + x) * 3;
        // Input Value
        uchar blue = input[idx];
        uchar green = input[idx + 1];
        uchar red = input[idx + 2];

        // RGB to YCrCB
        uchar Y = (uchar)(0.299f * red + 0.587f * green + 0.114f * blue);
        uchar Cb = (uchar)(128.0f + (blue - Y) * 0.564f);
        uchar Cr = (uchar)(128.0f + (red - Y) * 0.713f);

        // Output Value
        output[idx]     = Y;
        output[idx + 1] = Cb;
        output[idx + 2] = Cr;
    }
}

Mat ParallelYCrCBCUDA(Mat &input) {
    // Image size
    int rows = input.rows;
    int cols = input.cols;

    // Input and output data
    size_t dataSize = rows * cols * 3 * sizeof(uchar);
    uchar *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);

    // Copy input data to device
    hipMemcpy(d_input, input.data, dataSize, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    yCrCBKernel<<<gridSize, blockSize>>>(d_input, d_output, rows, cols);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy output data back to host
    Mat result(input.size(), CV_8UC3);
    hipMemcpy(result.data, d_output, dataSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return result;
}
